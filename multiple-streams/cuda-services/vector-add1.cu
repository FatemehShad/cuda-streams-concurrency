
#include <hip/hip_runtime.h>
#include <stdio.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

namespace vectorAdd1{
__global__ void vectorAdd1(float *a, float *b, float *c, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  c[i] = a[i] + b[i];
}

float maxError(float *a, int n) 
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

float* launchKernel()
{
  const int blockSize = 256;
  const int nStreams = 8;
  const int n = 1 << 20; // 1M elements
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);

  float *a, *b, *c, *d_a, *d_b, *d_c;

  //ALlocate pinned host memory
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) );      
  checkCuda( hipHostMalloc((void**)&b, bytes, hipHostMallocDefault) );      
  checkCuda( hipHostMalloc((void**)&c, bytes, hipHostMallocDefault) );      
  
  //Allocate device memory
  checkCuda( hipMalloc((void**)&d_a, bytes) ); 
  checkCuda( hipMalloc((void**)&d_b, bytes) ); 
  checkCuda( hipMalloc((void**)&d_c, bytes) );

  // float ms; // elapsed time in milliseconds
  
  //Setup CUDA Stream
  hipStream_t stream[nStreams];
  for (int i = 0; i < nStreams; ++i){
    checkCuda( hipStreamCreate(&stream[i]) );
  }

  // Initialize host arrays
  for(int i=0; i < n; i++){
    a[i] =10*i;
    b[i]=40*i;
  }

  /*
    Async: loop over copy from host to device, kernel invocation, and transfer data back from device to the host
  */
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset], 
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );

    checkCuda( hipMemcpyAsync(&d_b[offset], &a[offset], 
                              streamBytes, hipMemcpyHostToDevice, 
                              stream[i]) );
  }

  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    vectorAdd1<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, d_b, d_c, offset);
  }
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&c[offset], &d_c[offset], 
      streamBytes, hipMemcpyDeviceToHost,
      stream[i]) );
  }

  printf("Max error: %e\n", maxError(c, n));

  //Free up memory
  for (int i = 0; i < nStreams; ++i){
    checkCuda( hipStreamDestroy(stream[i]));
  }
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipHostFree(a);
  hipHostFree(b);

  return c;
  }
}